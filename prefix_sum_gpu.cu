#include <iostream>
#include <hip/hip_runtime.h>

#define BSIZE 1024

__global__ void prefix_sum_intra_block(int* d_input, int* d_output, int n) {
    __shared__ int temp[BSIZE];

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    // cargar datos en shared memory
    if (index < n) {
        temp[tid] = d_input[index];
    } else {
        temp[tid] = 0;
    }
    __syncthreads();

    //prefix sum en memoria compartida
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        int temp_val = (tid >= offset) ? temp[tid - offset] : 0;
        __syncthreads();
        if (tid >= offset) {
            temp[tid] += temp_val;
        }
        __syncthreads();
    }

    // escribir resultados en memoria global
    if (index < n) {
        d_output[index] = temp[tid];
    }
}

// calcular las sumas finales de cada bloque
__global__ void block_sums_kernel(int* d_output, int* d_block_sums, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n && threadIdx.x == blockDim.x - 1) {
        d_block_sums[blockIdx.x] = d_output[index];
    }
}

// ajustar las sumas entre bloques
__global__ void adjust_inter_block(int* d_output, int* d_block_sums, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (blockIdx.x > 0 && index < n) {
        d_output[index] += d_block_sums[blockIdx.x - 1];
    }
}

void prefix_sum_gpu(int* input, int* output, int n) {
    int threads_per_block = BSIZE;                                      // threads x bloque
    int grid_size = (n + threads_per_block - 1) / threads_per_block;    // bloques necesarios
    int *d_input, *d_output, *d_block_sums;

    hipMalloc((void**)&d_input, n * sizeof(int));
    hipMalloc((void**)&d_output, n * sizeof(int));
    hipMalloc((void**)&d_block_sums, grid_size * sizeof(int));

    hipMemcpy(d_input, input, n * sizeof(int), hipMemcpyHostToDevice);

    // prefix sum en cada bloque
    prefix_sum_intra_block<<<grid_size, threads_per_block>>>(d_input, d_output, n);
    hipDeviceSynchronize();

    // sumas finales de cada bloque
    block_sums_kernel<<<grid_size, threads_per_block>>>(d_output, d_block_sums, n);
    hipDeviceSynchronize();

    // copiar las sumas de bloques al host para ajuste inter-bloque
    int* h_block_sums = new int[grid_size];
    hipMemcpy(h_block_sums, d_block_sums, grid_size * sizeof(int), hipMemcpyDeviceToHost);

    // calcular los ajustes acumulativos de los bloques en el host
    for (int i = 1; i < grid_size; ++i) {
        h_block_sums[i] += h_block_sums[i - 1];
    }

    // copiar los ajustes acumulativos de vuelta a la GPU
    hipMemcpy(d_block_sums, h_block_sums, grid_size * sizeof(int), hipMemcpyHostToDevice);
    delete[] h_block_sums;

    // ajustar las sumas inter-bloque
    adjust_inter_block<<<grid_size, threads_per_block>>>(d_output, d_block_sums, n);
    hipDeviceSynchronize();

    // copiar el resultado final al host
    hipMemcpy(output, d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    // liberar memoria
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_block_sums);
}
